#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <string>
#include <vector>
#include <random>
#include <exception>
#include<unistd.h>
#include "mpcd_macro.cuh"
//#include "nonslipXperiodic.cuh"
#include "LEBC.cuh"
#include "reduction_sum.cuh"
#include "thermostat.cuh"
#include "streaming.cuh"
#include "noslip_MPCD_streaming.cuh"
#include "collision.cuh"
#include "gallileain_inv.cuh"
#include "rerstart_file.cuh"
#include "md_analyser.cuh"
#include "gpu_md.cuh"
#include "Active_gpu_md.cuh"
#include "noslip_MD_streaming.cuh"
#include "begining.cuh"
#include "logging.cuh"
#include <ctime>
#include "center_of_mass.cuh"
#include "reducefileC.cuh"



int main(int argc, const char* argv[])
{

    //I change this (argc!=16) to argc!=18 . because I added 2 other inputs which are "Activity" and "random_flag".
    // update: now I want to add another input which is boudary condition or BC, so I change argc to 19.
    std::cout<<argc<<"\n";
    if( argc !=19)
    {
        std::cout<<argc<<"\n";
        std::cout<<"Argument parsing failed!\n";
        std::string exeName = argv[0];
        std::cout<<exeName<<"\n";
        std::cout<<"Number of given arguments: "<<argc<<"\n";
        return 1;
    }
    std::string inputfile= argv[1]; //restart file name
    std::string basename = argv[2]; //output base name
    L[0] = atof(argv[3]); //dimension of the simulation in x direction
    L[1] = atof(argv[4]); //dimension of the simulation in y direction
    L[2] = atof(argv[5]); //dimension of the simulation in z direction
    density = atoi(argv[6]); //density of the particles
    n_md = atoi(argv[7]); //number of rings
    m_md = atof(argv[8]); //number of monomer in each ring
    shear_rate = atof(argv[9]); //shear rate
    h_md = atof(argv[10]); //md time step
    h_mpcd = atof(argv[11]); //mpcd time step
    swapsize = atoi(argv[12]);//output interval
    simuationtime = atoi(argv[13]);//final simulation step count
    TIME = atoi(argv[14]);//starting
    topology = atoi(argv[15]);
    Activity = atoi(argv[16]);//I added a parameter called activity which is either 0 or 1 ( either we have activity or we don't)
    random_flag = atoi(argv[17]);//a flag to see if we have random activity or not
    BC = atoi(argv[18]); //a parameter called BC which is ether 1 or 2 or 3( either we have periodic BC or no-slip BC Or combined.)
      
    double ux = shear_rate * L[2];
    double u_scale = 1.0;
    double DR = 0.001; //Rotational friction coefficient
    double Rh = 1.0;
    double delta_ratio = 0.33; 
    double *gama_T;
    gama_T = (double*) malloc(sizeof(double));
    *gama_T = 0.8;
    

    double *temperature;
    temperature = (double*) malloc(sizeof(double));
    *temperature = 1.0;
    double Pe = 10.0; //peclet number
    double l_eq = 1.0; // equilibrium length
    u_scale = Pe * l_eq *DR ; 
    int Nc = L[0]*L[1]*L[2]; //number of cells 
    int N =density* Nc; //number of particles
    int Nmd = n_md * m_md;//total number of monomers
     int grid_size = ((N + blockSize) / blockSize);
    int shared_mem_size = 3 * blockSize * sizeof(double); // allocate shared memory for the intermediate reduction results.
    
     //random generator
     hiprandGenerator_t gen;
     hiprandCreateGenerator(&gen, 
         HIPRAND_RNG_PSEUDO_DEFAULT);
     /* Set seed */
     hiprandSetPseudoRandomGeneratorSeed(gen, 
         4294967296ULL^time(NULL));
     hiprandState *devStates;
     hipMalloc((void **)&devStates, blockSize * grid_size *sizeof(hiprandState));
     setup_kernel<<<grid_size, blockSize>>>(time(NULL), devStates);
    

    // Allocate device memory for mpcd particle:
    double *d_x, *d_vx , *d_y , *d_vy , *d_z , *d_vz;
    int *d_index;
    hipMalloc((void**)&d_x, sizeof(double) * N);   hipMalloc((void**)&d_y, sizeof(double) * N);   hipMalloc((void**)&d_z, sizeof(double) * N);
    hipMalloc((void**)&d_vx, sizeof(double) * N);  hipMalloc((void**)&d_vy, sizeof(double) * N);  hipMalloc((void**)&d_vz, sizeof(double) * N);
    hipMalloc((void**)&d_index, sizeof(int) *N);

    //Allocate device memory for reduced mpcd files:
    int skipfactor = 3000;
    double *scalefactor;
    scalefactor = (double*) malloc(sizeof(double));
    *scalefactor = 1.0;
    int NN = int (N/skipfactor);

    double *d_xx; double *d_yy; double *d_zz;
    double *d_xx_lim1; double *d_yy_lim1; double *d_zz_lim1;
    double *d_xx_lim2; double *d_yy_lim2; double *d_zz_lim2;
    double *d_xx_lim3; double *d_yy_lim3; double *d_zz_lim3;
    double *d_xx_lim4; double *d_yy_lim4; double *d_zz_lim4;
    double *d_xx_lim5; double *d_yy_lim5; double *d_zz_lim5;
    double *d_xx_lim6; double *d_yy_lim6; double *d_zz_lim6;
    double *d_xx_lim7; double *d_yy_lim7; double *d_zz_lim7;
    hipMalloc((void**)&d_xx,sizeof(double)*NN); hipMalloc((void**)&d_yy,sizeof(double)*NN); hipMalloc((void**)&d_zz,sizeof(double)*NN);
    hipMalloc((void**)&d_xx_lim1,sizeof(double)*NN); hipMalloc((void**)&d_yy_lim1,sizeof(double)*NN); hipMalloc((void**)&d_zz_lim1,sizeof(double)*NN);
    hipMalloc((void**)&d_xx_lim2,sizeof(double)*NN); hipMalloc((void**)&d_yy_lim2,sizeof(double)*NN); hipMalloc((void**)&d_zz_lim2,sizeof(double)*NN);
    hipMalloc((void**)&d_xx_lim3,sizeof(double)*NN); hipMalloc((void**)&d_yy_lim3,sizeof(double)*NN); hipMalloc((void**)&d_zz_lim3,sizeof(double)*NN);
    hipMalloc((void**)&d_xx_lim4,sizeof(double)*NN); hipMalloc((void**)&d_yy_lim4,sizeof(double)*NN); hipMalloc((void**)&d_zz_lim4,sizeof(double)*NN);
    hipMalloc((void**)&d_xx_lim5,sizeof(double)*NN); hipMalloc((void**)&d_yy_lim5,sizeof(double)*NN); hipMalloc((void**)&d_zz_lim5,sizeof(double)*NN);
    hipMalloc((void**)&d_xx_lim6,sizeof(double)*NN); hipMalloc((void**)&d_yy_lim6,sizeof(double)*NN); hipMalloc((void**)&d_zz_lim6,sizeof(double)*NN);
    hipMalloc((void**)&d_xx_lim7,sizeof(double)*NN); hipMalloc((void**)&d_yy_lim7,sizeof(double)*NN); hipMalloc((void**)&d_zz_lim7,sizeof(double)*NN);
    double *d_endp_x; double *d_endp_y; double *d_endp_z;
    hipMalloc((void**)&d_endp_x,sizeof(double)*NN); hipMalloc((void**)&d_endp_y,sizeof(double)*NN); hipMalloc((void**)&d_endp_z,sizeof(double)*NN);
    
    //int decimalPlaces = 3; // Number of decimal places to keep
    double *roundedNumber_x; double *roundedNumber_y; double *roundedNumber_z;
    hipMalloc((void**)&roundedNumber_x, sizeof(double) *N);
    hipMalloc((void**)&roundedNumber_y, sizeof(double) *N);
    hipMalloc((void**)&roundedNumber_z, sizeof(double) *N);


    //Allocate device memory for reduced mpcd velocity files:
    double *d_vxx; double *d_vyy; double *d_vzz;
    double *d_vxx_lim1; double *d_vyy_lim1; double *d_vzz_lim1;
    double *d_vxx_lim2; double *d_vyy_lim2; double *d_vzz_lim2;
    double *d_vxx_lim3; double *d_vyy_lim3; double *d_vzz_lim3;
    double *d_vxx_lim4; double *d_vyy_lim4; double *d_vzz_lim4;
    double *d_vxx_lim5; double *d_vyy_lim5; double *d_vzz_lim5;
    double *d_vxx_lim6; double *d_vyy_lim6; double *d_vzz_lim6;
    double *d_vxx_lim7; double *d_vyy_lim7; double *d_vzz_lim7;
    hipMalloc((void**)&d_vxx,sizeof(double)*NN); hipMalloc((void**)&d_vyy,sizeof(double)*NN); hipMalloc((void**)&d_vzz,sizeof(double)*NN);
    hipMalloc((void**)&d_vxx_lim1,sizeof(double)*NN); hipMalloc((void**)&d_vyy_lim1,sizeof(double)*NN); hipMalloc((void**)&d_vzz_lim1,sizeof(double)*NN);
    hipMalloc((void**)&d_vxx_lim2,sizeof(double)*NN); hipMalloc((void**)&d_vyy_lim2,sizeof(double)*NN); hipMalloc((void**)&d_vzz_lim2,sizeof(double)*NN);
    hipMalloc((void**)&d_vxx_lim3,sizeof(double)*NN); hipMalloc((void**)&d_vyy_lim3,sizeof(double)*NN); hipMalloc((void**)&d_vzz_lim3,sizeof(double)*NN);
    hipMalloc((void**)&d_vxx_lim4,sizeof(double)*NN); hipMalloc((void**)&d_vyy_lim4,sizeof(double)*NN); hipMalloc((void**)&d_vzz_lim4,sizeof(double)*NN);
    hipMalloc((void**)&d_vxx_lim5,sizeof(double)*NN); hipMalloc((void**)&d_vyy_lim5,sizeof(double)*NN); hipMalloc((void**)&d_vzz_lim5,sizeof(double)*NN);
    hipMalloc((void**)&d_vxx_lim6,sizeof(double)*NN); hipMalloc((void**)&d_vyy_lim6,sizeof(double)*NN); hipMalloc((void**)&d_vzz_lim6,sizeof(double)*NN);
    hipMalloc((void**)&d_vxx_lim7,sizeof(double)*NN); hipMalloc((void**)&d_vyy_lim7,sizeof(double)*NN); hipMalloc((void**)&d_vzz_lim7,sizeof(double)*NN);
    
    //int decimalPlacess = 3; // Number of decimal places to keep
    double *roundedNumber_vx; double *roundedNumber_vy; double *roundedNumber_vz;
    hipMalloc((void**)&roundedNumber_vx, sizeof(double) *N);
    hipMalloc((void**)&roundedNumber_vy, sizeof(double) *N);
    hipMalloc((void**)&roundedNumber_vz, sizeof(double) *N);

    
    //Allocate device memory for box attributes:
    double *d_L, *d_r;   
    hipMalloc((void**)&d_L, sizeof(double) *3);
    hipMalloc((void**)&d_r, sizeof(double) *3);
    
    // Allocate device memory for cells:
    double *d_ux , *d_uy , *d_uz;
    int  *d_n;
    double *d_m;
    hipMalloc((void**)&d_ux, sizeof(double) * Nc); hipMalloc((void**)&d_uy, sizeof(double) * Nc); hipMalloc((void**)&d_uz, sizeof(double) * Nc);
    hipMalloc((void**)&d_n, sizeof(int) * Nc);     hipMalloc((void**)&d_m, sizeof(double) * Nc);
    //Allocate device memory for rotating angles and matrix:
    double *d_phi , *d_theta,*d_rot;
    hipMalloc((void**)&d_phi, sizeof(double) * Nc);    hipMalloc((void**)&d_theta , sizeof(double) *Nc);  hipMalloc((void**)&d_rot, sizeof(double) * Nc *9);

    int *dn_tot;
    hipMalloc((void**)&dn_tot, sizeof(int));
    double *N_avg;
    hipMalloc((void**)&N_avg, sizeof(double));
 
    int *sumblock_n;
    hipMalloc((void**)&sumblock_n, sizeof(int) * grid_size);

    double *dm_tot;
    hipMalloc((void**)&dm_tot, sizeof(double));
    double *M_avg;
    hipMalloc((void**)&M_avg, sizeof(double));

    
    double *sumblock_m;
    hipMalloc((void**)&sumblock_m, sizeof(double) * grid_size);

    double *a_x, *a_y, *a_z;
    hipMalloc((void**)&a_x, sizeof(double) * Nc); hipMalloc((void**)&a_y, sizeof(double) * Nc); hipMalloc((void**)&a_y, sizeof(double) * Nc);

    double* d_variance;
    hipMalloc((void**)&d_variance,  sizeof(double) * Nc);
    
 

    hiprandState *d_States;
    hipMalloc((void**)&d_States, sizeof(hiprandState) * Nc);

    //Allocate device memory for cell level thermostat atributes:
    double* d_e, *d_scalefactor;
    hipMalloc((void**)&d_e , sizeof(double) * Nc);
    hipMalloc((void**)&d_scalefactor , sizeof(double) * Nc);

    //Allocate device memory for md particle:
    double *d_mdX, *d_mdY, *d_mdZ, *d_mdVx, *d_mdVy , *d_mdVz, *d_mdAx , *d_mdAy, *d_mdAz;
    int *d_mdIndex;
    hipMalloc((void**)&d_mdX, sizeof(double) * Nmd);    hipMalloc((void**)&d_mdY, sizeof(double) * Nmd);    hipMalloc((void**)&d_mdZ, sizeof(double) * Nmd);
    hipMalloc((void**)&d_mdVx, sizeof(double) * Nmd);   hipMalloc((void**)&d_mdVy, sizeof(double) * Nmd);   hipMalloc((void**)&d_mdVz, sizeof(double) * Nmd);
    hipMalloc((void**)&d_mdAx, sizeof(double) * Nmd);   hipMalloc((void**)&d_mdAy, sizeof(double) * Nmd);   hipMalloc((void**)&d_mdAz, sizeof(double) * Nmd);
    hipMalloc((void**)&d_mdIndex, sizeof(int) * Nmd);
    ///////////////NEW MD attributes:
    double *md_Fx_holder , *md_Fy_holder , *md_Fz_holder;
    hipMalloc((void**)&md_Fx_holder, sizeof(double) * Nmd *(Nmd ));    hipMalloc((void**)&md_Fy_holder, sizeof(double) * Nmd *(Nmd ));    hipMalloc((void**)&md_Fz_holder, sizeof(double) * Nmd *(Nmd));
    

    //Allocate device memory for active and backward forces exerted on each MD particle:
    double *d_fa_kx , *d_fa_ky , *d_fa_kz , *d_fb_kx , *d_fb_ky , *d_fb_kz;
    hipMalloc((void**)&d_fa_kx, sizeof(double) * Nmd);    hipMalloc((void**)&d_fa_ky, sizeof(double) * Nmd);    hipMalloc((void**)&d_fa_kz, sizeof(double) * Nmd);
    hipMalloc((void**)&d_fb_kx, sizeof(double) * Nmd);    hipMalloc((void**)&d_fb_ky, sizeof(double) * Nmd);    hipMalloc((void**)&d_fb_kz, sizeof(double) * Nmd);
    
    
    //Allocate device memory for total active and backward forces:
    double *h_fa_x , *h_fa_y , *h_fa_z ;
    double *h_fb_x , *h_fb_y , *h_fb_z ;
    // hipMalloc((void**)&h_fa_x, sizeof(double)); hipMalloc((void**)&h_fa_y, sizeof(double)); hipMalloc((void**)&h_fa_z, sizeof(double));
    // hipMalloc((void**)&h_fb_x, sizeof(double)); hipMalloc((void**)&h_fb_y, sizeof(double)); hipMalloc((void**)&h_fb_z, sizeof(double));
    h_fa_x = (double*) malloc(sizeof(double)); h_fa_y = (double*) malloc(sizeof(double)); h_fa_z = (double*) malloc(sizeof(double));
    h_fb_x = (double*) malloc(sizeof(double)); h_fb_y = (double*) malloc(sizeof(double)); h_fb_z = (double*) malloc(sizeof(double));
    *h_fa_x=0.0; *h_fa_y=0.0; *h_fa_z=0.0;
    *h_fb_x=0.0; *h_fb_y=0.0; *h_fb_z=0.0;

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    //center of mass attributes:
    double *mdX_tot , *mdY_tot, *mdZ_tot ;
    double *dX_tot , *dY_tot, *dZ_tot ;
    mdX_tot = (double*) malloc(sizeof(double)); mdY_tot = (double*) malloc(sizeof(double)); mdZ_tot = (double*) malloc(sizeof(double));
    dX_tot = (double*) malloc(sizeof(double)); dY_tot = (double*) malloc(sizeof(double)); dZ_tot = (double*) malloc(sizeof(double));
    *mdX_tot=0.0; *mdY_tot=0.0; *mdZ_tot=0.0;
    *dX_tot=0.0; *dY_tot=0.0; *dZ_tot=0.0;

    

/////////////////////////////////////////////// I'd maximize the performance by adjusting new grid_size_ amd blockSize_ this way:
    int device = 0; // GPU device number (you can change this)
    hipSetDevice(device);

    int blockSize_;  // To store the recommended block size
    blockSize_ = 256; 
    int minGridSize; // To store the minimum grid size

    int dataSize = N; // Adjust this to your problem size
    int* d_data; // Device pointer for data array

    // Allocate memory on the GPU
    hipMalloc((void**)&d_data, dataSize * sizeof(int));

    // Determine the maximum potential block size
    //hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_, reduce_kernel, 0, dataSize);

    // Print the recommended block size
    std::cout << "Recommended Block Size: " << blockSize_ << std::endl;
    printf ("blocksize=%i", blockSize_);

    // Calculate the grid size based on your data size and the block size
    int grid_size_ = (dataSize + blockSize_ - 1) / blockSize_;
/////////////////////////////////////////////////////////////////////

    //allocate memory for counting zero factors in reducing and limiting the data to a specific box around the MD particles. 

    int *zerofactorsumblock; //an array to sum over all zero blocks. 
    int *zerofactorsumblock1, *zerofactorsumblock2, *zerofactorsumblock3, *zerofactorsumblock4, *zerofactorsumblock5, *zerofactorsumblock6, *zerofactorsumblock7;
    hipMalloc((void**)&zerofactorsumblock, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorsumblock1, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorsumblock2, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorsumblock3, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorsumblock4, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorsumblock5, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorsumblock6, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorsumblock7, sizeof(int) * grid_size_); 
    int *zerofactor; //a 0/1 array 
    int *zerofactor1,*zerofactor2, *zerofactor3, *zerofactor4, *zerofactor5, *zerofactor6, *zerofactor7;
    hipMalloc((void**)&zerofactor, sizeof(int) * N);
    hipMalloc((void**)&zerofactor1, sizeof(int) * N);
    hipMalloc((void**)&zerofactor2, sizeof(int) * N);
    hipMalloc((void**)&zerofactor3, sizeof(int) * N);
    hipMalloc((void**)&zerofactor4, sizeof(int) * N);
    hipMalloc((void**)&zerofactor5, sizeof(int) * N);
    hipMalloc((void**)&zerofactor6, sizeof(int) * N);
    hipMalloc((void**)&zerofactor7, sizeof(int) * N);
    int *zerofactorrsumblock; //an array to sum over all zero blocks.
    int *zerofactorrsumblock1,*zerofactorrsumblock2, *zerofactorrsumblock3, *zerofactorrsumblock4, *zerofactorrsumblock5, *zerofactorrsumblock6, *zerofactorrsumblock7; 
    hipMalloc((void**)&zerofactorrsumblock, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorrsumblock1, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorrsumblock2, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorrsumblock3, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorrsumblock4, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorrsumblock5, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorrsumblock6, sizeof(int) * grid_size_);
    hipMalloc((void**)&zerofactorrsumblock7, sizeof(int) * grid_size_);
    int *zerofactorr; //a 0/1 array
    int *zerofactorr1, *zerofactorr2, *zerofactorr3, *zerofactorr4, *zerofactorr5, *zerofactorr6, *zerofactorr7;
    hipMalloc((void**)&zerofactorr, sizeof(int) * N);
    hipMalloc((void**)&zerofactorr1, sizeof(int) * N);
    hipMalloc((void**)&zerofactorr2, sizeof(int) * N);
    hipMalloc((void**)&zerofactorr3, sizeof(int) * N);
    hipMalloc((void**)&zerofactorr4, sizeof(int) * N);
    hipMalloc((void**)&zerofactorr5, sizeof(int) * N);
    hipMalloc((void**)&zerofactorr6, sizeof(int) * N);
    hipMalloc((void**)&zerofactorr7, sizeof(int) * N);

//////////////////////////////////////////////////////////////////////

    double *CMsumblock_x; double *CMsumblock_y; double *CMsumblock_z;
    double *CMsumblock_mdx; double *CMsumblock_mdy; double *CMsumblock_mdz;

    hipMalloc((void**)&CMsumblock_x, grid_size_ * sizeof(double)); hipMalloc((void**)&CMsumblock_y, grid_size_ * sizeof(double)); hipMalloc((void**)&CMsumblock_z, grid_size_ * sizeof(double));
    hipMalloc((void**)&CMsumblock_mdx, grid_size * sizeof(double)); hipMalloc((void**)&CMsumblock_mdy, grid_size * sizeof(double)); hipMalloc((void**)&CMsumblock_mdz, grid_size * sizeof(double));

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    double *h_Xcm , *h_Ycm, *h_Zcm ; 
    //h_Xcm = (double*) malloc(sizeof(double)); h_Ycm = (double*) malloc(sizeof(double)); h_Zcm = (double*) malloc(sizeof(double));
    hipMalloc((void**)&h_Xcm, sizeof(double)); hipMalloc((void**)&h_Ycm, sizeof(double)); hipMalloc((void**)&h_Zcm, sizeof(double));
    
    //Allocate device memory for active and backward accelerations exerted on each MD particle:
   
    double *d_Aa_kx , *d_Aa_ky , *d_Aa_kz , *d_Ab_kx , *d_Ab_ky , *d_Ab_kz;
    hipMalloc((void**)&d_Aa_kx, sizeof(double) * Nmd);    hipMalloc((void**)&d_Aa_ky, sizeof(double) * Nmd);    hipMalloc((void**)&d_Aa_kz, sizeof(double) * Nmd);
    hipMalloc((void**)&d_Ab_kx, sizeof(double) * Nmd);    hipMalloc((void**)&d_Ab_ky, sizeof(double) * Nmd);    hipMalloc((void**)&d_Ab_kz, sizeof(double) * Nmd);



    //Allocate device memory for total active and backward accelerations:
    //host memory:
    double h_Aa_x , h_Aa_y, h_Aa_z;
    double h_Ab_x , h_Ab_y, h_Ab_z;
   
    
    //Allocate device memory for total d_Ax_tot, d_Ay_tot and d_Az_tot:
    double *d_Ax_tot , *d_Ay_tot , *d_Az_tot;
    hipMalloc((void**)&d_Ax_tot, sizeof(double)* Nmd);    hipMalloc((void**)&d_Ay_tot, sizeof(double)* Nmd);    hipMalloc((void**)&d_Az_tot, sizeof(double)* Nmd);

    //Allocate device memory for tanfential vectors ex , ey and ez:
    double *d_ex , *d_ey , *d_ez;
    hipMalloc((void**)&d_ex, sizeof(double) * Nmd);    hipMalloc((void**)&d_ey, sizeof(double) * Nmd);    hipMalloc((void**)&d_ez, sizeof(double) * Nmd);
    
    //Allocate device memory for block sum of ex , ey and ez:
    double *d_block_sum_ex , *d_block_sum_ey , *d_block_sum_ez;
    hipMalloc((void**)&d_block_sum_ex, sizeof(double) * grid_size);    hipMalloc((void**)&d_block_sum_ey, sizeof(double) * grid_size);    hipMalloc((void**)&d_block_sum_ez, sizeof(double) * grid_size);

    //Allocate device memory for random array:
    int *d_random_array;
    hipMalloc((void**)&d_random_array, sizeof(int) * Nmd);

    int *d_flag_array;
    hipMalloc((void**)&d_flag_array, sizeof(int) * Nmd);

    unsigned int d_seed;
    //is this seed correct?
    d_seed = (unsigned int)(time(NULL));

    //Allocate device memory for no slip boundary condition attributes:
   

    double  *d_x_o , *d_y_o, *d_z_o; //the x, y and z of the point where a mpcd particle crosses the box walls.
    hipMalloc((void**)&d_x_o, sizeof(double) * N);  hipMalloc((void**)&d_y_o, sizeof(double) * N);  hipMalloc((void**)&d_z_o, sizeof(double) * N);
    double  *d_Vx_o , *d_Vy_o, *d_Vz_o; //the VELOCITY of the point where a mpcd particle crosses the box walls.
    hipMalloc((void**)&d_Vx_o, sizeof(double) * N);  hipMalloc((void**)&d_Vy_o, sizeof(double) * N);  hipMalloc((void**)&d_Vz_o, sizeof(double) * N);
    double *totalT;
    hipMalloc((void**)&totalT, sizeof(double) * N);


    double  *d_mdX_o , *d_mdY_o, *d_mdZ_o;//the x, y and z of the point where a md particle crosses the box walls.
    hipMalloc((void**)&d_mdX_o, sizeof(double) * Nmd);  hipMalloc((void**)&d_mdY_o, sizeof(double) * Nmd);  hipMalloc((void**)&d_mdZ_o, sizeof(double) * Nmd);
    double  *d_mdVx_o , *d_mdVy_o, *d_mdVz_o;//the x, y and z of the point where a md particle crosses the box walls.
    hipMalloc((void**)&d_mdVx_o, sizeof(double) * Nmd);  hipMalloc((void**)&d_mdVy_o, sizeof(double) * Nmd);  hipMalloc((void**)&d_mdVz_o, sizeof(double) * Nmd);



    double *d_dt_min , *d_dt_x, *d_dt_y, *d_dt_z;//crossing point with box for MPCD paricles
    hipMalloc((void**)&d_dt_min, sizeof(double) * N);  hipMalloc((void**)&d_dt_x, sizeof(double) * N);  hipMalloc((void**)&d_dt_y, sizeof(double) * N);  hipMalloc((void**)&d_dt_z, sizeof(double) * N);
    

    double *d_md_dt_min, *d_md_dt_x, *d_md_dt_y, *d_md_dt_z;//crossing point with box for MD paricles
    hipMalloc((void**)&d_md_dt_min, sizeof(double) * Nmd);  hipMalloc((void**)&d_md_dt_x, sizeof(double) * Nmd);  hipMalloc((void**)&d_md_dt_y, sizeof(double) * Nmd);  hipMalloc((void**)&d_md_dt_z, sizeof(double) * Nmd);

    double *d_x_wall_dist, *d_y_wall_dist, *d_z_wall_dist, *d_wall_sign_x, *d_wall_sign_y, *d_wall_sign_z;
    hipMalloc((void**)&d_x_wall_dist, sizeof(double) * N);  hipMalloc((void**)&d_y_wall_dist, sizeof(double) * N);  hipMalloc((void**)&d_z_wall_dist, sizeof(double) * N);
    hipMalloc((void**)&d_wall_sign_x, sizeof(double) * N);  hipMalloc((void**)&d_wall_sign_y, sizeof(double) * N);  hipMalloc((void**)&d_wall_sign_z, sizeof(double) * N);


    double *d_mdX_wall_dist, *d_mdY_wall_dist, *d_mdZ_wall_dist, *d_wall_sign_mdX, *d_wall_sign_mdY, *d_wall_sign_mdZ;
    hipMalloc((void**)&d_mdX_wall_dist, sizeof(double) * Nmd);  hipMalloc((void**)&d_mdY_wall_dist, sizeof(double) * Nmd);  hipMalloc((void**)&d_mdZ_wall_dist, sizeof(double) * Nmd);
    hipMalloc((void**)&d_wall_sign_mdX, sizeof(double) * Nmd);  hipMalloc((void**)&d_wall_sign_mdY, sizeof(double) * Nmd);  hipMalloc((void**)&d_wall_sign_mdZ, sizeof(double) * Nmd);    
    ///////////////////////////////////////////////////////////////////////////////////

    if(Activity==0 && BC == 1){
        if (TIME ==0)start_simulation(basename, simuationtime , swapsize ,d_L, d_mdX , d_mdY , d_mdZ,
        d_mdVx , d_mdVy , d_mdVz , d_mdAx , d_mdAy , d_mdAz , md_Fx_holder, md_Fy_holder,md_Fz_holder,
        d_x , d_y , d_z , d_vx , d_vy , d_vz, gen , grid_size, totalT);
        else restarting_simulation(basename , inputfile , simuationtime , swapsize ,d_L, d_mdX , d_mdY , d_mdZ,
        d_mdVx , d_mdVy , d_mdVz , d_mdAx , d_mdAy , d_mdAz , md_Fx_holder, md_Fy_holder,md_Fz_holder,
        d_x , d_y , d_z , d_vx , d_vy , d_vz, ux , N , Nmd , TIME , grid_size);
    
        
        double real_time = TIME;
        int T =simuationtime/swapsize +TIME/swapsize;
        int delta = h_mpcd / h_md;
        xyz_trj(basename + "_traj.xyz", d_mdX, d_mdY , d_mdZ, Nmd);
        for(int t = TIME/swapsize ; t<T; t++)
        {
            for (int i =0;i<int(swapsize/h_mpcd); i++)
            {
                hiprandGenerateUniformDouble(gen, d_phi, Nc);
                hiprandGenerateUniformDouble(gen, d_theta, Nc);
                hiprandGenerateUniformDouble(gen, d_r, 3);

                

                MPCD_streaming(d_x , d_y , d_z , d_vx , d_vy , d_vz , h_mpcd , N , grid_size);
            

                MD_streaming(d_mdX , d_mdY , d_mdZ , d_mdVx , d_mdVy , d_mdVz ,
                    d_mdAx , d_mdAy , d_mdAz ,md_Fx_holder, md_Fy_holder, md_Fz_holder,
                    h_md , Nmd , density , d_L , ux , grid_size, delta,real_time);

                Sort_begin(d_x , d_y , d_z , d_vx, d_vy, d_vz, d_index , d_mdX , d_mdY , d_mdZ ,
                    d_mdVx, d_mdVy, d_mdVz, d_mdIndex ,ux , d_L , d_r , N , Nmd , real_time, grid_size);

                MPCD_MD_collision(d_vx , d_vy , d_vz , d_index,
                    d_mdVx , d_mdVy , d_mdVz , d_mdIndex,
                    d_ux , d_uy , d_uz , d_e , d_scalefactor , d_n , d_m ,
                    d_rot , d_theta , d_phi , N , Nmd ,Nc ,devStates , grid_size);
            
                Sort_finish(d_x , d_y , d_z , d_vx, d_vy, d_vz, d_index , 
                    d_mdX , d_mdY, d_mdZ, d_mdVx, d_mdVy, d_mdVz, d_mdIndex, ux, 
                    d_L , d_r , N , Nmd , real_time, grid_size);
            
                real_time += h_mpcd;
                 

            }
            double *temperature;
            logging(basename + "_log.log" , (t+1)*swapsize , d_mdVx , d_mdVy , d_mdVz , d_vx, d_vy , d_vz, N , Nmd, grid_size , temperature);
            xyz_trj(basename + "_traj.xyz", d_mdX, d_mdY , d_mdZ, Nmd);
            xyz_trj(basename + "_vel.xyz", d_mdVx, d_mdVy , d_mdVz, Nmd);
       
        }

        md_write_restart_file(basename, d_mdX , d_mdY , d_mdZ , d_mdVx , d_mdVy , d_mdVz , Nmd);
        mpcd_write_restart_file(basename ,d_x , d_y , d_z , d_vx , d_vy , d_vz , N);

    

    
        hipFree(d_x); hipFree(d_y); hipFree(d_z);
        hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);
        hipFree(d_ux); hipFree(d_uy); hipFree(d_uz);
        hipFree(d_rot); hipFree(d_phi); hipFree(d_theta);
        hipFree(devStates); hipFree(d_e); hipFree(d_scalefactor);
        //Free memory MD particles:
        hipFree(d_mdX);    hipFree(d_mdY);    hipFree(d_mdZ);
        hipFree(d_mdVx);   hipFree(d_mdVy);   hipFree(d_mdVz);
        hipFree(d_mdAx);   hipFree(d_mdAy);   hipFree(d_mdAz);
        hipFree(md_Fx_holder); hipFree(md_Fy_holder); hipFree(md_Fz_holder);
        hiprandDestroyGenerator(gen);

        std::cout<<"The program has terminated succesffuly at time:"<<real_time<<std::endl;
    }

    //this part is specific for when we don't have activity and no-slip boundary condition.
    else if (Activity==0 && BC ==2){

        if (TIME ==0)start_simulation(basename, simuationtime , swapsize ,d_L, d_mdX , d_mdY , d_mdZ,
        d_mdVx , d_mdVy , d_mdVz , d_mdAx , d_mdAy , d_mdAz , md_Fx_holder, md_Fy_holder,md_Fz_holder,
        d_x , d_y , d_z , d_vx , d_vy , d_vz, gen , grid_size, totalT);
        else restarting_simulation(basename , inputfile , simuationtime , swapsize ,d_L, d_mdX , d_mdY , d_mdZ,
        d_mdVx , d_mdVy , d_mdVz , d_mdAx , d_mdAy , d_mdAz , md_Fx_holder, md_Fy_holder,md_Fz_holder,
        d_x , d_y , d_z , d_vx , d_vy , d_vz, ux , N , Nmd , TIME , grid_size);
    
        
        double real_time = TIME;
        int T =simuationtime/swapsize +TIME/swapsize;
        int delta = h_mpcd / h_md;
        xyz_trj(basename + "_traj.xyz", d_mdX, d_mdY , d_mdZ, Nmd);
        for(int t = TIME/swapsize ; t<T; t++)
        {
            for (int i =0;i<int(swapsize/h_mpcd); i++)
            {
                hiprandGenerateUniformDouble(gen, d_phi, Nc);
                hiprandGenerateUniformDouble(gen, d_theta, Nc);
                hiprandGenerateUniformDouble(gen, d_r, 3);

                

                noslip_MPCD_streaming(d_x , d_y , d_z , d_vx , d_vy , d_vz , h_mpcd , N , grid_size, d_L, d_dt_x, d_dt_y, d_dt_z, d_dt_min, d_x_o, d_y_o, d_z_o, d_Vx_o , d_Vy_o , d_Vz_o, d_x_wall_dist, d_y_wall_dist, d_z_wall_dist, d_wall_sign_x, d_wall_sign_y, d_wall_sign_z, totalT);
            

                noslip_MD_streaming(d_mdX , d_mdY , d_mdZ , d_mdVx , d_mdVy , d_mdVz ,
                    d_mdAx , d_mdAy , d_mdAz ,md_Fx_holder, md_Fy_holder, md_Fz_holder,
                    h_md , Nmd , density , d_L , ux , grid_size, delta, real_time,
                    d_md_dt_min, d_md_dt_x, d_md_dt_y, d_md_dt_z, d_mdX_o, d_mdY_o, d_mdZ_o, d_mdVx_o, d_mdVy_o, d_mdVz_o, d_mdX_wall_dist, d_mdY_wall_dist, d_mdZ_wall_dist, d_wall_sign_mdX, d_wall_sign_mdY, d_wall_sign_mdZ );

                noslip_Sort_begin(d_x , d_y , d_z , d_vx, d_vy, d_vz, d_index , d_mdX , d_mdY , d_mdZ ,
                    d_mdVx, d_mdVy, d_mdVz, d_mdIndex ,ux , d_L , d_r , N , Nmd , real_time, grid_size);

                noslip_MPCD_MD_collision(d_vx , d_vy , d_vz , d_index,
                    d_mdVx , d_mdVy , d_mdVz , d_mdIndex,
                    d_ux , d_uy , d_uz , d_e , d_scalefactor , d_n , d_m ,
                    d_rot , d_theta , d_phi , N , Nmd ,Nc ,devStates , grid_size, dn_tot, N_avg, sumblock_n, dm_tot, M_avg, sumblock_m,
                    a_x, a_y, a_z, d_variance, d_States);
            
                noslip_Sort_finish(d_x , d_y , d_z , d_vx, d_vy, d_vz, d_index , 
                    d_mdX , d_mdY, d_mdZ, d_mdVx, d_mdVy, d_mdVz, d_mdIndex, ux, 
                    d_L , d_r , N , Nmd , real_time, grid_size);
            
                real_time += h_mpcd;
                 

            }
            double *temperature;
            logging(basename + "_log.log" , (t+1)*swapsize , d_mdVx , d_mdVy , d_mdVz , d_vx, d_vy , d_vz, N , Nmd, grid_size , temperature);
            xyz_trj(basename + "_traj.xyz", d_mdX, d_mdY , d_mdZ, Nmd);
            xyz_trj(basename + "_vel.xyz", d_mdVx, d_mdVy , d_mdVz, Nmd);
       
        }

        md_write_restart_file(basename, d_mdX , d_mdY , d_mdZ , d_mdVx , d_mdVy , d_mdVz , Nmd);
        mpcd_write_restart_file(basename ,d_x , d_y , d_z , d_vx , d_vy , d_vz , N);

    

    
        hipFree(d_x); hipFree(d_y); hipFree(d_z);
        hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);
        hipFree(d_ux); hipFree(d_uy); hipFree(d_uz);
        hipFree(d_rot); hipFree(d_phi); hipFree(d_theta);
        hipFree(devStates); hipFree(d_e); hipFree(d_scalefactor);
        //Free memory MD particles:
        hipFree(d_mdX);    hipFree(d_mdY);    hipFree(d_mdZ);
        hipFree(d_mdVx);   hipFree(d_mdVy);   hipFree(d_mdVz);
        hipFree(d_mdAx);   hipFree(d_mdAy);   hipFree(d_mdAz);
        hipFree(md_Fx_holder); hipFree(md_Fy_holder); hipFree(md_Fz_holder);
        hipFree(d_x_o); hipFree(d_y_o); hipFree(d_z_o);
        hipFree(d_Vx_o); hipFree(d_Vy_o); hipFree(d_Vz_o);
        hipFree(d_mdX_o); hipFree(d_mdY_o); hipFree(d_mdZ_o);
        hipFree(d_mdVx_o); hipFree(d_mdVy_o); hipFree(d_mdVz_o);
        hipFree(d_dt_min); hipFree(d_md_dt_min);
        hipFree(d_dt_x); hipFree(d_md_dt_x);
        hipFree(d_dt_y); hipFree(d_md_dt_y);
        hipFree(d_dt_z); hipFree(d_md_dt_z);
        hipFree(d_x_wall_dist); hipFree(d_y_wall_dist); hipFree(d_z_wall_dist);
        hipFree(d_wall_sign_x); hipFree(d_wall_sign_y); hipFree(d_wall_sign_z);
        hipFree(d_mdX_wall_dist); hipFree(d_mdY_wall_dist); hipFree(d_mdZ_wall_dist);
        hipFree(d_wall_sign_mdX); hipFree(d_wall_sign_mdY); hipFree(d_wall_sign_mdZ);
        hipFree(totalT);
        hiprandDestroyGenerator(gen);

        std::cout<<"The program has terminated succesffuly at time:"<<real_time<<std::endl;

    }
}

/*
    //this part is specific for when we have activity and periodic boundary condition. 
    else if (Activity==1 && BC==1){ 

        double real_time = TIME;

        double temper0 = 1;
        //temperature = temp_calc(d_vx, d_vy , d_vz , d_mdVx , d_mdVy , d_mdVz , density, N , Nmd, grid_size);
        
        //gama_T= temper0 / ((2*Rh)*(2*Rh)*DR*delta_ratio) ; 
        *gama_T = 0.8;

      
        if (TIME ==0) Active_start_simulation(basename, simuationtime , swapsize ,d_L, d_mdX , d_mdY , d_mdZ,
        d_mdVx , d_mdVy , d_mdVz , d_mdAx , d_mdAy , d_mdAz , md_Fx_holder, md_Fy_holder,md_Fz_holder,
        d_x , d_y , d_z , d_vx , d_vy , d_vz, d_fa_kx,d_fa_ky, d_fa_kz, d_fb_kx, d_fb_ky, d_fb_kz, d_Aa_kx, d_Aa_ky, d_Aa_kz, d_Ab_kx, d_Ab_ky, d_Ab_kz,d_Ax_tot, d_Ay_tot, d_Az_tot, d_ex, d_ey,d_ez, h_fa_x, h_fa_y, h_fa_z, h_fb_x, h_fb_y, h_fb_z, d_block_sum_ex, d_block_sum_ey, d_block_sum_ez, gen , grid_size, real_time, gama_T, d_random_array, d_seed, d_flag_array, u_scale);
        else Active_restarting_simulation(basename , inputfile , simuationtime , swapsize ,d_L, d_mdX , d_mdY , d_mdZ,
        d_mdVx , d_mdVy , d_mdVz , d_mdAx , d_mdAy , d_mdAz , md_Fx_holder, md_Fy_holder,md_Fz_holder,
        d_x , d_y , d_z , d_vx , d_vy , d_vz, d_fa_kx, d_fa_ky, d_fa_kz, d_fb_kx, d_fb_ky, d_fb_kz, d_Aa_kx, d_Aa_ky, d_Aa_kz, d_Ab_kx, d_Ab_ky, d_Ab_kz,d_Ax_tot,d_Ay_tot, d_Az_tot, d_ex, d_ey, d_ez, h_fa_x, h_fa_y, h_fa_z, h_fb_x, h_fb_y, h_fb_z, d_block_sum_ex, d_block_sum_ey, d_block_sum_ez, ux , N , Nmd , TIME , grid_size, real_time, gama_T, d_random_array, d_seed, d_flag_array, u_scale);
    
        
        //double real_time = TIME;
        int T =simuationtime/swapsize +TIME/swapsize;
        int delta = h_mpcd / h_md;
      
        xyz_trj(basename + "_traj.xyz", d_mdX, d_mdY , d_mdZ, Nmd);
        //xyz_trj(basename + "_mpcdtraj.xyz", d_x, d_y , d_z, N);
        reducetraj(basename, d_x, d_y , d_z, d_xx, d_yy, d_zz, d_vx, d_vy, d_vz, d_vxx, d_vyy, d_vzz, N, skipfactor, grid_size, roundedNumber_x, roundedNumber_y, roundedNumber_z, zerofactorr, roundedNumber_vx, roundedNumber_vy, roundedNumber_vz, zerofactor, zerofactorrsumblock, blockSize_, grid_size_,
        d_xx_lim1, d_yy_lim1, d_zz_lim1, zerofactorr1,
        d_xx_lim2, d_yy_lim2, d_zz_lim2, zerofactorr2,
        d_xx_lim3, d_yy_lim3, d_zz_lim3, zerofactorr3,
        d_xx_lim4, d_yy_lim4, d_zz_lim4, zerofactorr4,
        d_xx_lim5, d_yy_lim5, d_zz_lim5, zerofactorr5,
        d_xx_lim6, d_yy_lim6, d_zz_lim6, zerofactorr6,
        d_xx_lim7, d_yy_lim7, d_zz_lim7, zerofactorr7,
        d_vxx_lim1, d_vyy_lim1, d_vzz_lim1,
        d_vxx_lim2, d_vyy_lim2, d_vzz_lim2, 
        d_vxx_lim3, d_vyy_lim3, d_vzz_lim3, 
        d_vxx_lim4, d_vyy_lim4, d_vzz_lim4, 
        d_vxx_lim5, d_vyy_lim5, d_vzz_lim5,
        d_vxx_lim6, d_vyy_lim6, d_vzz_lim6,  
        d_vxx_lim7, d_vyy_lim7, d_vzz_lim7, 
        zerofactorrsumblock1,zerofactorrsumblock2,zerofactorrsumblock3,zerofactorrsumblock4,zerofactorrsumblock5,zerofactorrsumblock6,zerofactorrsumblock7);

 
        for(int t = TIME/swapsize ; t<T; t++) //T is TIME/swapsize + simulationtime/swapsize. it goes 10 steps from TIME/swapsize till end (in case swapsize is 100 and siumlationtime is 1000). 
        {
            
            for (int i =0;i<int(swapsize/h_mpcd); i++) //swapsize/h_mpcd when swapsize is 100 and h_mpcd is 0.1, is equal to 1000.
            {
                
                hiprandGenerateUniformDouble(gen, d_phi, Nc);
                hiprandGenerateUniformDouble(gen, d_theta, Nc);
                hiprandGenerateUniformDouble(gen, d_r, 3);
                



           

                //double temperature;
                //temperature = 0.0;
                //temperature = temp_calc(d_vx, d_vy , d_vz , d_mdVx , d_mdVy , d_mdVz , density, N , Nmd, grid_size);
                //printf("T=%lf\n",temperature);
                //double *gama_T=2.0;
                //*gama_T= temperature / ((2*Rh)*(2*Rh)*DR*delta_ratio) ; 
                //printf("gama_T=%lf\n",*gama_T);

                //go to center of mass reference frame:
                CM_system(d_mdX, d_mdY, d_mdZ,d_x, d_y, d_z, Nmd, N, mdX_tot, mdY_tot, mdZ_tot, dX_tot, dY_tot, dZ_tot, grid_size, shared_mem_size, blockSize_, grid_size_, density, 1, h_Xcm, h_Ycm, h_Zcm, CMsumblock_x, CMsumblock_y, CMsumblock_z, CMsumblock_mdx, CMsumblock_mdy, CMsumblock_mdz, topology );

                Active_MPCD_streaming(d_x , d_y , d_z , d_vx , d_vy , d_vz ,h_mpcd ,N ,grid_size ,
                 h_fa_x ,h_fa_y ,h_fa_z ,h_fb_x ,h_fb_y ,h_fb_z ,d_ex ,d_ey , d_ez, d_block_sum_ex ,d_block_sum_ey ,d_block_sum_ez ,
                 L ,Nmd ,ux , density ,1 ,real_time ,m_md , topology, shared_mem_size);
            

                Active_MD_streaming(d_mdX , d_mdY , d_mdZ , d_mdVx , d_mdVy , d_mdVz ,
                    d_mdAx , d_mdAy , d_mdAz ,md_Fx_holder, md_Fy_holder, md_Fz_holder, d_fa_kx, d_fa_ky, d_fa_kz, d_fb_kx, d_fb_ky, d_fa_kz, 
                    d_Aa_kx, d_Aa_ky, d_Aa_kz, d_Ab_kx, d_Ab_ky, d_Ab_kz, d_Ax_tot, d_Ay_tot, d_Az_tot, d_ex, d_ey, d_ez,
                    h_fa_x, h_fa_y, h_fa_z, h_fb_x, h_fb_y, h_fb_z, d_block_sum_ex, d_block_sum_ey, d_block_sum_ez, 
                    h_md , Nmd ,density ,d_L ,ux ,grid_size ,delta ,real_time ,m_md ,N ,density ,1 , gama_T, d_random_array, d_seed, topology, h_Xcm, h_Ycm, h_Zcm, d_flag_array, u_scale);
                
                Sort_begin(d_x , d_y , d_z ,d_vx, d_vy, d_vz, d_index , d_mdX , d_mdY , d_mdZ ,
                    d_mdVx, d_mdVy, d_mdVz, d_mdIndex ,ux , d_L , d_r , N , Nmd , real_time, grid_size);

                MPCD_MD_collision(d_vx , d_vy , d_vz , d_index,
                    d_mdVx , d_mdVy , d_mdVz , d_mdIndex,
                    d_ux , d_uy , d_uz , d_e , d_scalefactor , d_n , d_m ,
                    d_rot , d_theta , d_phi , N , Nmd ,Nc ,devStates , grid_size);
            
                Sort_finish(d_x , d_y , d_z , d_vx, d_vy, d_vz, d_index , 
                    d_mdX , d_mdY , d_mdZ, d_mdVx, d_mdVy, d_mdVz, d_mdIndex, ux, 
                    d_L , d_r , N , Nmd , real_time, grid_size);
            
                real_time += h_mpcd;
                 

            }
            
            
            logging(basename + "_log.log" , (t+1)*swapsize , d_mdVx , d_mdVy , d_mdVz , d_vx, d_vy , d_vz, N , Nmd, grid_size, temperature );
           
            //printf("T=%f\n",*temperature);
            *gama_T= (*temperature) / ((2*Rh)*(2*Rh)*DR*delta_ratio) ; /////problem is here?

            xyz_trj(basename + "_traj.xyz", d_mdX, d_mdY , d_mdZ, Nmd);
            xyz_trj(basename + "_vel.xyz", d_mdVx, d_mdVy , d_mdVz, Nmd);
            reducetraj(basename, d_x, d_y , d_z, d_xx, d_yy, d_zz, d_vx, d_vy, d_vz, d_vxx, d_vyy, d_vzz, N, skipfactor, grid_size, roundedNumber_x, roundedNumber_y, roundedNumber_z, zerofactorr,roundedNumber_vx, roundedNumber_vy, roundedNumber_vz, zerofactor, zerofactorrsumblock, blockSize_, grid_size_,
                d_xx_lim1,  d_yy_lim1,  d_zz_lim1, zerofactorr1,
                d_xx_lim2,  d_yy_lim2,  d_zz_lim2, zerofactorr2,
                d_xx_lim3,  d_yy_lim3,  d_zz_lim3, zerofactorr3,
                d_xx_lim4,  d_yy_lim4,  d_zz_lim4, zerofactorr4,
                d_xx_lim5,  d_yy_lim5,  d_zz_lim5, zerofactorr5,
                d_xx_lim6,  d_yy_lim6,  d_zz_lim6, zerofactorr6,
                d_xx_lim7,  d_yy_lim7,  d_zz_lim7, zerofactorr7,
                d_vxx_lim1, d_vyy_lim1, d_vzz_lim1,
                d_vxx_lim2, d_vyy_lim2, d_vzz_lim2, 
                d_vxx_lim3, d_vyy_lim3, d_vzz_lim3, 
                d_vxx_lim4, d_vyy_lim4, d_vzz_lim4, 
                d_vxx_lim5, d_vyy_lim5, d_vzz_lim5,
                d_vxx_lim6, d_vyy_lim6, d_vzz_lim6,  
                d_vxx_lim7, d_vyy_lim7, d_vzz_lim7, 
                zerofactorrsumblock1,zerofactorrsumblock2,zerofactorrsumblock3,zerofactorrsumblock4,zerofactorrsumblock5,zerofactorrsumblock6,zerofactorrsumblock7);
            reducevel(basename, d_vx, d_vy, d_vz, d_vxx, d_vyy, d_vzz, d_x, d_y, d_z, N, skipfactor, grid_size,roundedNumber_vx, roundedNumber_vy, roundedNumber_vz, zerofactor, zerofactorsumblock, blockSize_, grid_size_,
                d_vxx_lim1, d_vyy_lim1, d_vzz_lim1, zerofactor1,
                d_vxx_lim2, d_vyy_lim2, d_vzz_lim2,  zerofactor2,
                d_vxx_lim3, d_vyy_lim3, d_vzz_lim3, zerofactor3,
                d_vxx_lim4, d_vyy_lim4, d_vzz_lim4,  zerofactor4,
                d_vxx_lim5, d_vyy_lim5, d_vzz_lim5, zerofactor5,
                d_vxx_lim6, d_vyy_lim6, d_vzz_lim6,  zerofactor6,
                d_vxx_lim7, d_vyy_lim7, d_vzz_lim7,  zerofactor7,
                zerofactorsumblock1 ,zerofactorsumblock2 ,zerofactorsumblock3 ,zerofactorsumblock4 ,zerofactorsumblock5 ,zerofactorsumblock6 ,zerofactorsumblock7);
            xyz_veltraj_both(basename, d_xx, d_yy, d_zz,d_vxx, d_vyy, d_vzz, NN, d_endp_x, d_endp_y, d_endp_z, scalefactor, grid_size);
            //xyz_trj(basename + "_mpcdtraj.xyz", d_x, d_y , d_z, N);
            //xyz_trj(basename + "_mpcdvel.xyz", d_vx, d_vy , d_vz, N);

            
       
        }

        md_write_restart_file(basename, d_mdX , d_mdY , d_mdZ , d_mdVx , d_mdVy , d_mdVz , Nmd);
        mpcd_write_restart_file(basename ,d_x , d_y , d_z , d_vx , d_vy , d_vz , N);

    

    
        hipFree(d_x); hipFree(d_y); hipFree(d_z);
        hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);
        hipFree(d_ux); hipFree(d_uy); hipFree(d_uz);
        hipFree(d_rot); hipFree(d_phi); hipFree(d_theta);
        hipFree(devStates); hipFree(d_e); hipFree(d_scalefactor);
        //Free memory MD particles:
        hipFree(d_mdX);    hipFree(d_mdY);    hipFree(d_mdZ);
        hipFree(d_mdVx);   hipFree(d_mdVy);   hipFree(d_mdVz);
        hipFree(d_mdAx);   hipFree(d_mdAy);   hipFree(d_mdAz);
        hipFree(md_Fx_holder); hipFree(md_Fy_holder); hipFree(md_Fz_holder);
        hipFree(d_fa_kx); hipFree(d_fa_ky); hipFree(d_fa_kz);
        hipFree(d_fb_kx); hipFree(d_fb_ky); hipFree(d_fb_kz);
        hipFree(d_Aa_kx); hipFree(d_Aa_ky); hipFree(d_Aa_kz);
        hipFree(d_Ab_kx); hipFree(d_Ab_ky); hipFree(d_Ab_kz);
        hipFree(d_Ax_tot); hipFree(d_Ay_tot); hipFree(d_Az_tot);
        hipFree(d_ex); hipFree(d_ey); hipFree(d_ez);
        hipFree(d_block_sum_ex); hipFree(d_block_sum_ey); hipFree(d_block_sum_ez);
        hipFree(d_random_array);
        hipFree(d_L); hipFree(d_r); 
        hipFree(d_m); hipFree(d_n); 
        hipFree(d_index); hipFree(d_mdIndex);
        hipFree(h_Xcm); hipFree(h_Ycm); hipFree(h_Zcm);
        hipFree(CMsumblock_x); hipFree(CMsumblock_y); hipFree(CMsumblock_z);
        hipFree(CMsumblock_mdx); hipFree(CMsumblock_mdy); hipFree(CMsumblock_mdz);
        //hipFree(gama_T);
        hipFree(d_flag_array);
        hipFree(d_xx); hipFree(d_yy); hipFree(d_zz);
        hipFree(d_endp_x); hipFree(d_endp_y); hipFree(d_endp_z);
        hipFree(d_vxx); hipFree(d_vyy); hipFree(d_vzz);
        hipFree(roundedNumber_x); hipFree(roundedNumber_y); hipFree(roundedNumber_z);
        hipFree(roundedNumber_vx); hipFree(roundedNumber_vy); hipFree(roundedNumber_vz);
        hipFree(zerofactor);  hipFree(zerofactorr);
        hipFree(zerofactorsumblock); hipFree(zerofactorrsumblock);
        hiprandDestroyGenerator(gen);
        
        //reducefile_traj();
        //reducefile_vel();

        std::cout<<"The program has terminated succesffuly at time:"<<real_time<<std::endl;
    }



    else if (Activity==1 && BC==2){ 

        double real_time = TIME;

        double temper0 = 1;
        //temperature = temp_calc(d_vx, d_vy , d_vz , d_mdVx , d_mdVy , d_mdVz , density, N , Nmd, grid_size);
        
        //gama_T= temper0 / ((2*Rh)*(2*Rh)*DR*delta_ratio) ; 
        *gama_T = 0.8;

       
        if (TIME ==0) Active_start_simulation(basename, simuationtime , swapsize ,d_L, d_mdX , d_mdY , d_mdZ,
        d_mdVx , d_mdVy , d_mdVz , d_mdAx , d_mdAy , d_mdAz , md_Fx_holder, md_Fy_holder,md_Fz_holder,
        d_x , d_y , d_z , d_vx , d_vy , d_vz, d_fa_kx,d_fa_ky, d_fa_kz, d_fb_kx, d_fb_ky, d_fb_kz, d_Aa_kx, d_Aa_ky, d_Aa_kz, d_Ab_kx, d_Ab_ky, d_Ab_kz,d_Ax_tot, d_Ay_tot, d_Az_tot, d_ex, d_ey,d_ez, h_fa_x, h_fa_y, h_fa_z, h_fb_x, h_fb_y, h_fb_z, d_block_sum_ex, d_block_sum_ey, d_block_sum_ez, gen , grid_size, real_time, gama_T, d_random_array, d_seed, d_flag_array, u_scale);
        else Active_restarting_simulation(basename , inputfile , simuationtime , swapsize ,d_L, d_mdX , d_mdY , d_mdZ,
        d_mdVx , d_mdVy , d_mdVz , d_mdAx , d_mdAy , d_mdAz , md_Fx_holder, md_Fy_holder,md_Fz_holder,
        d_x , d_y , d_z , d_vx , d_vy , d_vz, d_fa_kx, d_fa_ky, d_fa_kz, d_fb_kx, d_fb_ky, d_fb_kz, d_Aa_kx, d_Aa_ky, d_Aa_kz, d_Ab_kx, d_Ab_ky, d_Ab_kz,d_Ax_tot,d_Ay_tot, d_Az_tot, d_ex, d_ey, d_ez, h_fa_x, h_fa_y, h_fa_z, h_fb_x, h_fb_y, h_fb_z, d_block_sum_ex, d_block_sum_ey, d_block_sum_ez, ux , N , Nmd , TIME , grid_size, real_time, gama_T, d_random_array, d_seed, d_flag_array, u_scale);
    
        
        //double real_time = TIME;
        int T =simuationtime/swapsize +TIME/swapsize;
        int delta = h_mpcd / h_md;
      
        xyz_trj(basename + "_traj.xyz", d_mdX, d_mdY , d_mdZ, Nmd);
        //xyz_trj(basename + "_mpcdtraj.xyz", d_x, d_y , d_z, N);
        reducetraj(basename, d_x, d_y , d_z, d_xx, d_yy, d_zz, d_vx, d_vy, d_vz, d_vxx, d_vyy, d_vzz, N, skipfactor, grid_size, roundedNumber_x, roundedNumber_y, roundedNumber_z, zerofactorr, roundedNumber_vx, roundedNumber_vy, roundedNumber_vz, zerofactor, zerofactorrsumblock, blockSize_, grid_size_,
        d_xx_lim1, d_yy_lim1, d_zz_lim1, zerofactorr1,
        d_xx_lim2, d_yy_lim2, d_zz_lim2, zerofactorr2,
        d_xx_lim3, d_yy_lim3, d_zz_lim3, zerofactorr3,
        d_xx_lim4, d_yy_lim4, d_zz_lim4, zerofactorr4,
        d_xx_lim5, d_yy_lim5, d_zz_lim5, zerofactorr5,
        d_xx_lim6, d_yy_lim6, d_zz_lim6, zerofactorr6,
        d_xx_lim7, d_yy_lim7, d_zz_lim7, zerofactorr7,
        d_vxx_lim1, d_vyy_lim1, d_vzz_lim1,
        d_vxx_lim2, d_vyy_lim2, d_vzz_lim2, 
        d_vxx_lim3, d_vyy_lim3, d_vzz_lim3, 
        d_vxx_lim4, d_vyy_lim4, d_vzz_lim4, 
        d_vxx_lim5, d_vyy_lim5, d_vzz_lim5,
        d_vxx_lim6, d_vyy_lim6, d_vzz_lim6,  
        d_vxx_lim7, d_vyy_lim7, d_vzz_lim7, 
        zerofactorrsumblock1,zerofactorrsumblock2,zerofactorrsumblock3,zerofactorrsumblock4,zerofactorrsumblock5,zerofactorrsumblock6,zerofactorrsumblock7);

 
        for(int t = TIME/swapsize ; t<T; t++) //T is TIME/swapsize + simulationtime/swapsize. it goes 10 steps from TIME/swapsize till end (in case swapsize is 100 and siumlationtime is 1000). 
        {
            
            for (int i =0;i<int(swapsize/h_mpcd); i++) //swapsize/h_mpcd when swapsize is 100 and h_mpcd is 0.1, is equal to 1000.
            {
                
                hiprandGenerateUniformDouble(gen, d_phi, Nc);
                hiprandGenerateUniformDouble(gen, d_theta, Nc);
                hiprandGenerateUniformDouble(gen, d_r, 3);
                



         

                //double temperature;
                //temperature = 0.0;
                //temperature = temp_calc(d_vx, d_vy , d_vz , d_mdVx , d_mdVy , d_mdVz , density, N , Nmd, grid_size);
                //printf("T=%lf\n",temperature);
                //double *gama_T=2.0;
                //*gama_T= temperature / ((2*Rh)*(2*Rh)*DR*delta_ratio) ; 
                //printf("gama_T=%lf\n",*gama_T);

                //go to center of mass reference frame:
                CM_system(d_mdX, d_mdY, d_mdZ,d_x, d_y, d_z, Nmd, N, mdX_tot, mdY_tot, mdZ_tot, dX_tot, dY_tot, dZ_tot, grid_size, shared_mem_size, blockSize_, grid_size_, density, 1, h_Xcm, h_Ycm, h_Zcm, CMsumblock_x, CMsumblock_y, CMsumblock_z, CMsumblock_mdx, CMsumblock_mdy, CMsumblock_mdz, topology );

                noslip_Active_MPCD_streaming(d_x , d_y , d_z , d_vx , d_vy , d_vz ,h_mpcd ,N ,grid_size ,
                 h_fa_x ,h_fa_y ,h_fa_z ,h_fb_x ,h_fb_y ,h_fb_z ,d_ex ,d_ey , d_ez, d_block_sum_ex ,d_block_sum_ey ,d_block_sum_ez ,
                 L ,Nmd ,ux , density ,1 ,real_time ,m_md , topology, shared_mem_size, d_L, d_x_latest, d_y_latest, d_z_latest, d_vx_latest, d_vy_latest, d_vz_latest,  d_dt1, d_x_o, d_y_o, d_z_o, d_x_wallDist_P, d_y_wallDist_P, d_z_wallDist_P, d_x_wallDist_N, d_y_wallDist_N, d_z_wallDist_N);
            

                noslip_Active_MD_streaming(d_mdX , d_mdY , d_mdZ , d_mdVx , d_mdVy , d_mdVz ,
                    d_mdAx , d_mdAy , d_mdAz ,md_Fx_holder, md_Fy_holder, md_Fz_holder, d_fa_kx, d_fa_ky, d_fa_kz, d_fb_kx, d_fb_ky, d_fa_kz, 
                    d_Aa_kx, d_Aa_ky, d_Aa_kz, d_Ab_kx, d_Ab_ky, d_Ab_kz, d_Ax_tot, d_Ay_tot, d_Az_tot, d_ex, d_ey, d_ez,
                    h_fa_x, h_fa_y, h_fa_z, h_fb_x, h_fb_y, h_fb_z, d_block_sum_ex, d_block_sum_ey, d_block_sum_ez, 
                    h_md , Nmd ,density ,d_L ,ux ,grid_size ,delta ,real_time ,m_md ,N ,density ,1 , gama_T, d_random_array, d_seed, topology, h_Xcm, h_Ycm, h_Zcm, d_flag_array, u_scale,
                    d_L, d_mdX_latest, d_mdY_latest, d_mdZ_latest,d_mdVx_latest, d_mdVy_latest, d_mdVz_latest, d_md_dt1, d_mdX_o, d_mdY_o, d_mdZ_o, d_mdX_wallDist_P, d_mdY_wallDist_P, d_mdZ_wallDist_P, d_mdX_wallDist_N, d_mdY_wallDist_N, d_mdZ_wallDist_N);
                
                noslip_Sort_begin(d_x , d_y , d_z ,d_vx, d_vy, d_vz, d_index , d_mdX , d_mdY , d_mdZ ,
                    d_mdVx, d_mdVy, d_mdVz, d_mdIndex ,ux , d_L , d_r , N , Nmd , real_time, grid_size);

                MPCD_MD_collision(d_vx , d_vy , d_vz , d_index,
                    d_mdVx , d_mdVy , d_mdVz , d_mdIndex,
                    d_ux , d_uy , d_uz , d_e , d_scalefactor , d_n , d_m ,
                    d_rot , d_theta , d_phi , N , Nmd ,Nc ,devStates , grid_size);
            
                noslip_Sort_finish(d_x , d_y , d_z , d_vx, d_vy, d_vz, d_index , 
                    d_mdX , d_mdY , d_mdZ, d_mdVx, d_mdVy, d_mdVz, d_mdIndex, ux, 
                    d_L , d_r , N , Nmd , real_time, grid_size);
            
                real_time += h_mpcd;
                 

            }
            
            
            logging(basename + "_log.log" , (t+1)*swapsize , d_mdVx , d_mdVy , d_mdVz , d_vx, d_vy , d_vz, N , Nmd, grid_size, temperature );
           
            //printf("T=%f\n",*temperature);
            *gama_T= (*temperature) / ((2*Rh)*(2*Rh)*DR*delta_ratio) ; /////problem is here?

            xyz_trj(basename + "_traj.xyz", d_mdX, d_mdY , d_mdZ, Nmd);
            xyz_trj(basename + "_vel.xyz", d_mdVx, d_mdVy , d_mdVz, Nmd);
            reducetraj(basename, d_x, d_y , d_z, d_xx, d_yy, d_zz, d_vx, d_vy, d_vz, d_vxx, d_vyy, d_vzz, N, skipfactor, grid_size, roundedNumber_x, roundedNumber_y, roundedNumber_z, zerofactorr,roundedNumber_vx, roundedNumber_vy, roundedNumber_vz, zerofactor, zerofactorrsumblock, blockSize_, grid_size_,
                d_xx_lim1,  d_yy_lim1,  d_zz_lim1, zerofactorr1,
                d_xx_lim2,  d_yy_lim2,  d_zz_lim2, zerofactorr2,
                d_xx_lim3,  d_yy_lim3,  d_zz_lim3, zerofactorr3,
                d_xx_lim4,  d_yy_lim4,  d_zz_lim4, zerofactorr4,
                d_xx_lim5,  d_yy_lim5,  d_zz_lim5, zerofactorr5,
                d_xx_lim6,  d_yy_lim6,  d_zz_lim6, zerofactorr6,
                d_xx_lim7,  d_yy_lim7,  d_zz_lim7, zerofactorr7,
                d_vxx_lim1, d_vyy_lim1, d_vzz_lim1,
                d_vxx_lim2, d_vyy_lim2, d_vzz_lim2, 
                d_vxx_lim3, d_vyy_lim3, d_vzz_lim3, 
                d_vxx_lim4, d_vyy_lim4, d_vzz_lim4, 
                d_vxx_lim5, d_vyy_lim5, d_vzz_lim5,
                d_vxx_lim6, d_vyy_lim6, d_vzz_lim6,  
                d_vxx_lim7, d_vyy_lim7, d_vzz_lim7, 
                zerofactorrsumblock1,zerofactorrsumblock2,zerofactorrsumblock3,zerofactorrsumblock4,zerofactorrsumblock5,zerofactorrsumblock6,zerofactorrsumblock7);
            reducevel(basename, d_vx, d_vy, d_vz, d_vxx, d_vyy, d_vzz, d_x, d_y, d_z, N, skipfactor, grid_size,roundedNumber_vx, roundedNumber_vy, roundedNumber_vz, zerofactor, zerofactorsumblock, blockSize_, grid_size_,
                d_vxx_lim1, d_vyy_lim1, d_vzz_lim1, zerofactor1,
                d_vxx_lim2, d_vyy_lim2, d_vzz_lim2,  zerofactor2,
                d_vxx_lim3, d_vyy_lim3, d_vzz_lim3, zerofactor3,
                d_vxx_lim4, d_vyy_lim4, d_vzz_lim4,  zerofactor4,
                d_vxx_lim5, d_vyy_lim5, d_vzz_lim5, zerofactor5,
                d_vxx_lim6, d_vyy_lim6, d_vzz_lim6,  zerofactor6,
                d_vxx_lim7, d_vyy_lim7, d_vzz_lim7,  zerofactor7,
                zerofactorsumblock1 ,zerofactorsumblock2 ,zerofactorsumblock3 ,zerofactorsumblock4 ,zerofactorsumblock5 ,zerofactorsumblock6 ,zerofactorsumblock7);
            xyz_veltraj_both(basename, d_xx, d_yy, d_zz,d_vxx, d_vyy, d_vzz, NN, d_endp_x, d_endp_y, d_endp_z, scalefactor, grid_size);
            //xyz_trj(basename + "_mpcdtraj.xyz", d_x, d_y , d_z, N);
            //xyz_trj(basename + "_mpcdvel.xyz", d_vx, d_vy , d_vz, N);

            
       
        }

        md_write_restart_file(basename, d_mdX , d_mdY , d_mdZ , d_mdVx , d_mdVy , d_mdVz , Nmd);
        mpcd_write_restart_file(basename ,d_x , d_y , d_z , d_vx , d_vy , d_vz , N);

    

    
        hipFree(d_x); hipFree(d_y); hipFree(d_z);
        hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);
        hipFree(d_ux); hipFree(d_uy); hipFree(d_uz);
        hipFree(d_rot); hipFree(d_phi); hipFree(d_theta);
        hipFree(devStates); hipFree(d_e); hipFree(d_scalefactor);
        //Free memory MD particles:
        hipFree(d_mdX);    hipFree(d_mdY);    hipFree(d_mdZ);
        hipFree(d_mdVx);   hipFree(d_mdVy);   hipFree(d_mdVz);
        hipFree(d_mdAx);   hipFree(d_mdAy);   hipFree(d_mdAz);
        hipFree(md_Fx_holder); hipFree(md_Fy_holder); hipFree(md_Fz_holder);
        hipFree(d_fa_kx); hipFree(d_fa_ky); hipFree(d_fa_kz);
        hipFree(d_fb_kx); hipFree(d_fb_ky); hipFree(d_fb_kz);
        hipFree(d_Aa_kx); hipFree(d_Aa_ky); hipFree(d_Aa_kz);
        hipFree(d_Ab_kx); hipFree(d_Ab_ky); hipFree(d_Ab_kz);
        hipFree(d_Ax_tot); hipFree(d_Ay_tot); hipFree(d_Az_tot);
        hipFree(d_ex); hipFree(d_ey); hipFree(d_ez);
        hipFree(d_block_sum_ex); hipFree(d_block_sum_ey); hipFree(d_block_sum_ez);
        hipFree(d_random_array);
        hipFree(d_L); hipFree(d_r); 
        hipFree(d_m); hipFree(d_n); 
        hipFree(d_index); hipFree(d_mdIndex);
        hipFree(h_Xcm); hipFree(h_Ycm); hipFree(h_Zcm);
        hipFree(CMsumblock_x); hipFree(CMsumblock_y); hipFree(CMsumblock_z);
        hipFree(CMsumblock_mdx); hipFree(CMsumblock_mdy); hipFree(CMsumblock_mdz);
        //hipFree(gama_T);
        hipFree(d_flag_array);
        hipFree(d_xx); hipFree(d_yy); hipFree(d_zz);
        hipFree(d_endp_x); hipFree(d_endp_y); hipFree(d_endp_z);
        hipFree(d_vxx); hipFree(d_vyy); hipFree(d_vzz);
        hipFree(roundedNumber_x); hipFree(roundedNumber_y); hipFree(roundedNumber_z);
        hipFree(roundedNumber_vx); hipFree(roundedNumber_vy); hipFree(roundedNumber_vz);
        hipFree(zerofactor);  hipFree(zerofactorr);
        hipFree(zerofactorsumblock); hipFree(zerofactorrsumblock);
        hipFree(d_x_latest); hipFree(d_y_latest); hipFree(d_z_latest); 
        hipFree(d_mdX_latest); hipFree(d_mdY_latest); hipFree(d_mdZ_latest);
        hipFree(d_vx_latest); hipFree(d_vy_latest); hipFree(d_vz_latest); 
        hipFree(d_mdVx_latest); hipFree(d_mdVy_latest); hipFree(d_mdVz_latest);
        hipFree(d_x_o); hipFree(d_y_o); hipFree(d_z_o);
        hipFree(d_mdX_o); hipFree(d_mdY_o); hipFree(d_mdZ_o);
        hipFree(d_dt1); hipFree(d_md_dt1);
        hipFree(d_x_wallDist_P); hipFree(d_y_wallDist_P); hipFree(d_z_wallDist_P);
        hipFree(d_x_wallDist_N); hipFree(d_y_wallDist_N); hipFree(d_z_wallDist_N);
        hipFree(d_mdX_wallDist_P); hipFree(d_mdY_wallDist_P); hipFree(d_mdZ_wallDist_P);
        hipFree(d_mdX_wallDist_N); hipFree(d_mdY_wallDist_N); hipFree(d_mdZ_wallDist_N);







        hiprandDestroyGenerator(gen);
        
        //reducefile_traj();
        //reducefile_vel();

        std::cout<<"The program has terminated succesffuly at time:"<<real_time<<std::endl;
    }

}

*/





